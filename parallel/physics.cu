#include "hip/hip_runtime.h"
#include <math.h>
#include <stdlib.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include "physics.h"
#include "functions.h"

/**
 * Function to find a free cell nearby if given starting cell is taken.
 * @param start start cell to search for another free cell
 * @param grid grid where the search is taken
 * @param return free cell near start
 */
__device__ static Cell* find_free_cell_nearby(Cell* start, Grid* grid)
{
    for (int i = -PROXIMITY_DISTANCE; i <= PROXIMITY_DISTANCE; i++)
    {
        for (int j = -PROXIMITY_DISTANCE; j <= PROXIMITY_DISTANCE; j++)
        {
            /**
             * Looking for a nearby position and checking if its free. In that case, that cell will be taken.
             */
            Vector position = {start->position.x + i, start->position.y + j};
            correct_position(&position, grid->size);
            Cell* cell = device_access_grid(grid, position);
            if (cell->type == FREE)
            {
                return cell;
            }
        }
    }
    return start;
}

__device__ void movement(Cell *cell, Grid *new_grid, hiprandState* rand_state)
{
    //If cell is free, ignore it.
    if (cell->type == FREE)
        return;
    
    /**
     * Computing box muller numbers for forces felt by the body and getting body mass.
     */
    float box_muller_number[2];
    box_muller(box_muller_number, rand_state);
    float mass = get_mass(cell->type);

    /**
     * Computing deltaV with Langevin equation and then updating cell's position and velocity.
     */
    Vector delta_velocity = 
    {
        .x = langevin_equation(cell->velocity.x, box_muller_number[0], mass),
        .y = langevin_equation(cell->velocity.y, box_muller_number[1], mass)
    };
    cell->velocity.x += delta_velocity.x;
    cell->velocity.y += delta_velocity.y;
    cell->position.x += round(cell->velocity.x * TIMESTEP);
    cell->position.y += round(cell->velocity.y * TIMESTEP);

    /**
     * Checking if the computed position is inside the grid and if it is free.
     */
    correct_position(&(cell->position), new_grid->size);
    Cell* new_cell = device_access_grid(new_grid, cell->position);
    if (new_cell->type != FREE)
    {
        new_cell = find_free_cell_nearby(new_cell, new_grid);
    }
    *new_cell = *cell;
}

__device__ float inline langevin_equation(float velocity, float collision_forces, float mass)
{
    return (-LAMBDA * velocity + collision_forces) / mass * TIMESTEP;
}

__device__ float inline get_mass(Type type)
{
    //Lymphocytes cells have a much higher mass than antigens and antibodies.
    switch (type)
    {
    case B:
    case T:
        return 0.2;
    case Ag:
    case Ab:
        return 0.01;
    }
}

__device__ void box_muller(float box_muller_number[2], hiprandState* rand_state)
{
    float 
        u1 = (float)(hiprand(rand_state)) / (float)(RAND_MAX),
        u2 = (float)(hiprand(rand_state)) / (float)(RAND_MAX);
    box_muller_number[0] = sqrt(-2 * log(u1)) * cos(2 * PI * u2);
    box_muller_number[1] = sqrt(-2 * log(u1)) * sin(2 * PI * u2);
}